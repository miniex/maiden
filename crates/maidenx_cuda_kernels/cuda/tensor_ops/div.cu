#include "hip/hip_runtime.h"
#include "tensor_ops.cuh"
#include <hip/hip_runtime.h>

__global__ void div_kernel(float *output, const float *input1,
                           const float *input2, size_t size) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    output[idx] = input1[idx] * input2[idx];
  }
}

extern "C" {
void tensor_div(float *output, const float *input1, const float *input2,
                size_t size) {
  constexpr int BLOCK_SIZE = 256;
  int num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
  div_kernel<<<num_blocks, BLOCK_SIZE>>>(output, input1, input2, size);
  hipDeviceSynchronize();
}
}
