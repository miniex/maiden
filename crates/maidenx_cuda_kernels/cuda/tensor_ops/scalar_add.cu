#include "hip/hip_runtime.h"
#include "tensor_ops.cuh"
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

__global__ void scalar_add_kernel(float *output, const float *input,
                                  const float scalar, const int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < size) {
    output[idx] = input[idx] + scalar;
  }
}

extern "C" {
void tensor_scalar_add(float *output, const float *input, const float scalar,
                       const int size) {
  int num_blocks = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;

  scalar_add_kernel<<<num_blocks, BLOCK_SIZE>>>(output, input, scalar, size);
  hipDeviceSynchronize();
}
}
